
#include <hip/hip_runtime.h>
#include <iostream>
#include <string>

int main(int argc, char *argv[]) {
  std::cout << "Gray-Scott CUDA implementation" << std::endl;

  double diffusion_rate_a = 0.0;
  double diffusion_rate_b = 0.0;
  double feed_rate = 0.0;
  double kill_rate = 0.0;
  int time_steps = 0;
  double delta_t = 0.0;
  std::string image_file = "";

  for (int i = 1; i < argc; ++i) {
    std::string arg = argv[i];
    if (arg == "--diffusion_rate_a") {
      diffusion_rate_a = std::stod(argv[++i]);
    } else if (arg == "--diffusion_rate_b") {
      diffusion_rate_b = std::stod(argv[++i]);
    } else if (arg == "--feed_rate") {
      feed_rate = std::stod(argv[++i]);
    } else if (arg == "--kill_rate") {
      kill_rate = std::stod(argv[++i]);
    } else if (arg == "--time_steps") {
      time_steps = std::stoi(argv[++i]);
    } else if (arg == "--delta_t") {
      delta_t = std::stod(argv[++i]);
    } else if (arg == "--image_file") {
      image_file = argv[++i];
    }
  }

  std::cout << "diffusion_rate_a: " << diffusion_rate_a << std::endl;
  std::cout << "diffusion_rate_b: " << diffusion_rate_b << std::endl;
  std::cout << "feed_rate: " << feed_rate << std::endl;
  std::cout << "kill_rate: " << kill_rate << std::endl;
  std::cout << "time_steps: " << time_steps << std::endl;
  std::cout << "delta_t: " << delta_t << std::endl;
  std::cout << "image_file: " << image_file << std::endl;

  return 0;
}