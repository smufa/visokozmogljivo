#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime_api.h"

#define STB_IMAGE_IMPLEMENTATION
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image.h"
#include "stb_image_write.h"

#define COLOR_CHANNELS 0

__global__ void copy_image(const unsigned char *imageIn, unsigned char *imageOut, const int width, const int height, const int cpp)
{

    int gidx = blockDim.x * blockIdx.x + threadIdx.x;
    int gidy = blockDim.y * blockIdx.y + threadIdx.y;
    if (gidx == 0 & gidy == 0)
    {
        printf("DEVICE: START COPY\n");
    }
    for (int i = gidx; i < height; i += blockDim.x * gridDim.x)
    {
        for (int j = gidy; j < width; j += blockDim.y * gridDim.y)
        {
            for (int c = 0; c < cpp; c += 1)
            {
                imageOut[(i * width + j) * cpp + c] = imageIn[(i * width + j) * cpp + c];
            }
        }
    }

}

int main(int argc, char *argv[])
{

    if (argc < 3)
    {
        printf("USAGE: sample input_image output_image\n");
        exit(EXIT_FAILURE);
    }

    char szImage_in_name[255];
    char szImage_out_name[255];

    snprintf(szImage_in_name, 255, "%s", argv[1]);
    snprintf(szImage_out_name, 255, "%s", argv[2]);

    // Load image from file and allocate space for the output image
    int width, height, cpp;
    unsigned char *h_imageIn = stbi_load(szImage_in_name, &width, &height, &cpp, COLOR_CHANNELS);

    if (h_imageIn == NULL)
    {
        printf("Error reading loading image %s!\n", szImage_in_name);
        exit(EXIT_FAILURE);
    }
    printf("Loaded image %s of size %dx%d.\n", szImage_in_name, width, height);
    const size_t datasize = width * height * cpp * sizeof(unsigned char);
    unsigned char *h_imageOut = (unsigned char *)malloc(datasize);

    // Setup Thread organization
    dim3 blockSize(16, 16);
    dim3 gridSize((height-1)/blockSize.x+1,(width-1)/blockSize.y+1);
    //dim3 gridSize(1, 1);

    unsigned char *d_imageIn;
    unsigned char *d_imageOut;

    // Allocate memory on the device
    checkCudaErrors(hipMalloc(&d_imageIn, datasize));
    checkCudaErrors(hipMalloc(&d_imageOut, datasize));

    // Use CUDA events to measure execution time
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Copy image to device and run kernel
    hipEventRecord(start);
    checkCudaErrors(hipMemcpy(d_imageIn, h_imageIn, datasize, hipMemcpyHostToDevice));
    copy_image<<<gridSize, blockSize>>>(d_imageIn, d_imageOut, width, height, cpp);
    checkCudaErrors(hipMemcpy(h_imageOut, d_imageOut, datasize, hipMemcpyDeviceToHost));
    getLastCudaError("copy_image() execution failed\n");
    hipEventRecord(stop);

    hipEventSynchronize(stop);

    // Print time
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Kernel Execution time is: %0.3f milliseconds \n", milliseconds);

    // Write the output file
    char szImage_out_name_temp[255];
    strncpy(szImage_out_name_temp, szImage_out_name, 255);
    char *token = strtok(szImage_out_name_temp, ".");
    char *FileType = NULL;
    while (token != NULL)
    {
        FileType = token;
        token = strtok(NULL, ".");
    }

    if (!strcmp(FileType, "png"))
        stbi_write_png(szImage_out_name, width, height, cpp, h_imageOut, width * cpp);
    else if (!strcmp(FileType, "jpg"))
        stbi_write_jpg(szImage_out_name, width, height, cpp, h_imageOut, 100);
    else if (!strcmp(FileType, "bmp"))
        stbi_write_bmp(szImage_out_name, width, height, cpp, h_imageOut);
    else
        printf("Error: Unknown image format %s! Only png, bmp, or bmp supported.\n", FileType);

    // Free device memory
    checkCudaErrors(hipFree(d_imageIn));
    checkCudaErrors(hipFree(d_imageOut));

    // Clean-up events
    hipEventDestroy(start);
    hipEventDestroy(stop);

    // Free host memory
    free(h_imageIn);
    free(h_imageOut);

    return 0;
}
